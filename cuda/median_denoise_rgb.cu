#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


typedef struct {
    int width;
    int height;
    unsigned char *data; // RGB data stored as [R, G, B, R, G, B, ...]
} PPMImage;

// Read PPM (P6 format)
PPMImage* read_ppm(const char *filename) {
    FILE *fp = fopen(filename, "rb");
    if (!fp) { perror("Error opening file"); return NULL; }

    PPMImage *img = (PPMImage*)malloc(sizeof(PPMImage));
    char version[3];
    if (fscanf(fp, "%2s", version) != 1) {
        fprintf(stderr, "Error reading PPM version\n");
        fclose(fp);
        free(img);
        return NULL;
    }
    if (version[1] != '6') {
        fprintf(stderr, "Only P6 supported\n");
        fclose(fp);
        free(img);
        return NULL;
    }

    if (fscanf(fp, "%d %d %*d", &img->width, &img->height) != 2) {
        fprintf(stderr, "Error reading image dimensions\n");
        fclose(fp);
        free(img);
        return NULL;
    }
    fgetc(fp); // Skip newline

    img->data = (unsigned char*)malloc(img->width * img->height * 3);
    if (fread(img->data, 1, img->width * img->height * 3, fp) != img->width * img->height * 3) {
        fprintf(stderr, "Error reading image data\n");
        fclose(fp);
        free(img->data);
        free(img);
        return NULL;
    }
    fclose(fp);
    return img;
}

// Write PPM (P6 format)
void write_ppm(const char *filename, PPMImage *img) {
    FILE *fp = fopen(filename, "wb");
    fprintf(fp, "P6\n%d %d\n255\n", img->width, img->height);
    fwrite(img->data, 1, img->width * img->height * 3, fp);
    fclose(fp);
}
__device__ void bubble_sort(unsigned char *window, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = i + 1; j < size; j++) {
            if (window[i] > window[j]) {
                unsigned char temp = window[i];
                window[i] = window[j];
                window[j] = temp;
            }
        }
    }
}

__global__ void median_filter_kernel(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        for (int c = 0; c < 3; c++) {
            unsigned char window[9];
            int idx = 0;

            for (int dy = -1; dy <= 1; dy++) {
                for (int dx = -1; dx <= 1; dx++) {
                    int neighbor_idx = ((y + dy) * width + (x + dx)) * 3 + c;
                    window[idx++] = input[neighbor_idx];
                }
            }

            bubble_sort(window, 9);
            int output_idx = (y * width + x) * 3 + c;
            output[output_idx] = window[4]; // Median
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <input.ppm> <output.ppm>\n", argv[0]);
        return 1;
    }

    const char *input_file = argv[1];
    const char *output_file = argv[2];

    // Start timing for the entire process
    clock_t total_start_time = clock();

    // Read input image
    PPMImage *input = read_ppm(input_file);
    if (!input) return 1;
    
    // Allocate output image
    PPMImage *output = (PPMImage*)malloc(sizeof(PPMImage));
    output->width = input->width;
    output->height = input->height;
    output->data = (unsigned char*)malloc(input->width * input->height * 3);

    
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, input->width * input->height * 3);
    hipMalloc(&d_output, input->width * input->height * 3);
    hipMemcpy(d_input, input->data, input->width * input->height * 3, hipMemcpyHostToDevice);
    
    dim3 threads_per_block(16, 16);
    dim3 blocks_per_grid((input->width + 15) / 16, (input->height + 15) / 16);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);
    median_filter_kernel<<<blocks_per_grid, threads_per_block>>>(d_input, d_output, input->width, input->height);
    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Median Kernel execution time: %.6f seconds.\n", milliseconds/1000);

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(output->data, d_output, input->width * input->height * 3, hipMemcpyDeviceToHost);

    // Write output image
    write_ppm(output_file, output);

    // End timing for the entire process
    clock_t total_end_time = clock();
    double total_elapsed_time = (double)(total_end_time - total_start_time) / CLOCKS_PER_SEC;
    printf("Total (Median) process completed in %.4f seconds.\n", total_elapsed_time);

    // Free memory
    free(input->data);
    free(input);
    free(output->data);
    free(output);
    
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
